#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <string>

#define checkCUDNN(expression)                                  \
{                                                               \
    hipdnnStatus_t status = (expression);                        \
    if (status != HIPDNN_STATUS_SUCCESS) {                       \
        std::cerr  << "Error on line " << __LINE__ << ": "      \
                   << hipdnnGetErrorString(status) << std::endl; \
        std::exit(EXIT_FAILURE);                                \
    }                                                           \
}

int setTensorDescriptor(hipdnnTensorDescriptor_t activationDesc,
                        const int numDim,
                        const long shape[]) {
    int batchSize = 0;
    int channels = 0;
    switch (numDim) {
        case 2:
            batchSize = shape[0];
            channels = shape[1];
            checkCUDNN(hipdnnSetTensor4dDescriptor(activationDesc,
                                                  HIPDNN_TENSOR_NCHW,
                                                  HIPDNN_DATA_FLOAT,
                                                  batchSize,
                                                  channels, 1, 1));
            break;

        case 4:
            batchSize = shape[0];
            channels = shape[1];
            int height = shape[2];
            int width = shape[3];
            checkCUDNN(hipdnnSetTensor4dDescriptor(activationDesc,
                                                  HIPDNN_TENSOR_NCHW,
                                                  HIPDNN_DATA_FLOAT,
                                                  batchSize,
                                                  channels,
                                                  height,
                                                  width));
            break;
            // TODO: handle other cases and errors

    }
    return 0;
}

int cudnnReLUForward(const DLArrayHandle input, DLArrayHandle output) {
    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;

    assert(input->shape[0] == output->shape[0]);
    assert(input->shape[1] == output->shape[1]);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnActivationDescriptor_t activation_descriptor;
    checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
    checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
                                            HIPDNN_ACTIVATION_RELU, // type of activation
                                            HIPDNN_PROPAGATE_NAN, // reluNanOpt
                                            0));  //relu_coef

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    setTensorDescriptor(input_descriptor, input->ndim, input->shape);

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    setTensorDescriptor(output_descriptor, output->ndim, output->shape);

    const float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnActivationForward(cudnn,
                                      activation_descriptor,
                                      &alpha,
                                      input_descriptor,
                                      input_data,
                                      &beta,
                                      output_descriptor,
                                      output_data));

    hipdnnDestroyActivationDescriptor(activation_descriptor);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);

    return 0;
}

int cudnnConv2DForward(const DLArrayHandle input,
                       const DLArrayHandle filter,
                       const DLArrayHandle bias,
                       const int stride_height,
                       const int stride_width,
                       const int padding_height,
                       const int padding_width,
                       DLArrayHandle output) {

    const int input_dim = input->ndim;
    const int output_dim = output->ndim;
    assert(input_dim == 4);
    assert(output_dim == 4);

    const int filter_shape = filter->ndim;
    assert(filter_shape == 4);
    const int num_filters = filter->shape[0];
    const int num_outputs = filter->shape[1];
    const int filter_height = filter->shape[2];
    const int filter_width = filter->shape[3];

    const int bias_dim = bias->ndim;
    assert(bias_dim == 1);
    assert(bias->shape[0] == num_filters);

    const float *input_data = (const float *) input->data;
    const float *filter_date = (const float *) filter->data;
    const float *bias_data = (const float *) bias->data;
    float *output_data = (float *) output->data;

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);


    // creating input and output tensors
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    setTensorDescriptor(input_descriptor, input->ndim, input->shape);

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    setTensorDescriptor(output_descriptor, output->ndim, output->shape);

    // create filter tensors
    hipdnnFilterDescriptor_t filter_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filter_descriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/num_outputs,
            /*in_channels=*/num_filters,
            /*kernel_height=*/filter_height,
            /*kernel_width=*/filter_width));
    // create convolution tensor
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
            /*pad_height=*/padding_height,
            /*pad_width=*/padding_width,
            /*vertical_stride=*/stride_height,
            /*horizontal_stride=*/stride_width,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
            /*computeType=*/HIPDNN_DATA_FLOAT));

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                                   input_descriptor,
                                                   filter_descriptor,
                                                   convolution_descriptor,
                                                   output_descriptor,
                                                   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
            /*memoryLimitInBytes=*/0,
                                                   &convolution_algorithm));

    size_t workspace_bytes{0};
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                       input_descriptor,
                                                       filter_descriptor,
                                                       convolution_descriptor,
                                                       output_descriptor,
                                                       convolution_algorithm,
                                                       &workspace_bytes));
    //std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB" << std::endl;
    assert(workspace_bytes > 0);

    void *d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    const float alpha = 1.0f, beta = 0.0f;

    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                       &alpha,
                                       input_descriptor,
                                       input_data,
                                       filter_descriptor,
                                       filter_date,
                                       convolution_descriptor,
                                       convolution_algorithm,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       output_descriptor,
                                       output_data));

    // adding bias tensor
    hipdnnTensorDescriptor_t bias_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&bias_descriptor));
    //setTensorDescriptor(bias_descriptor, bias->ndim, bias->shape);
    checkCUDNN(hipdnnSetTensor4dDescriptor(bias_descriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          1,
                                          num_filters,
                                          1,
                                          1));
    checkCUDNN(hipdnnAddTensor(cudnn,
                              &alpha,
                              bias_descriptor,
                              bias_data,
                              &alpha,
                              output_descriptor,
                              output_data));

    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);

    return 0;
}

int cudnnConv2DBackwardBias(const DLArrayHandle output_grads,
                            DLArrayHandle bias_grads) {

    const float *output_grads_data = (const float *) output_grads->data;
    float *bias_grads_data = (float *) bias_grads->data;

    const int bias_grads_dim = bias_grads->ndim;
    assert(bias_grads_dim == 1);
    const int num_filters = bias_grads->shape[0];

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // creating output_grads descriptor
    hipdnnTensorDescriptor_t output_grads_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_grads_descriptor));
    setTensorDescriptor(output_grads_descriptor, output_grads->ndim, output_grads->shape);

    // bias descriptor
    hipdnnTensorDescriptor_t bias_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&bias_descriptor));
    checkCUDNN(hipdnnSetTensor4dDescriptor(bias_descriptor,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          1,
                                          num_filters,
                                          1,
                                          1));

    const float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnConvolutionBackwardBias(cudnn,
                                            &alpha,
                                            output_grads_descriptor,
                                            output_grads_data,
                                            &beta,
                                            bias_descriptor,
                                            bias_grads_data
    ));

    hipdnnDestroyTensorDescriptor(bias_descriptor);
    hipdnnDestroyTensorDescriptor(output_grads_descriptor);
    hipdnnDestroy(cudnn);

    return 0;
}


int cudnnConv2DBackwardData(const DLArrayHandle filter,
                            const DLArrayHandle output_grads,
                            const int stride_height,
                            const int stride_width,
                            const int padding_height,
                            const int padding_width,
                            DLArrayHandle data_grad) {

    //const int input_dim = input->ndim;
    const int data_grad_dim = data_grad->ndim;
    //assert(input_dim == 4);
    assert(data_grad_dim == 4);

    const int filter_shape = filter->ndim;
    assert(filter_shape == 4);

    const int num_filters = filter->shape[0];
    const int num_outputs = filter->shape[1];
    const int filter_height = filter->shape[2];
    const int filter_width = filter->shape[3];

    //const float *input_data = (const float *) input->data;
    const float *filter_date = (const float *) filter->data;
    const float *output_grads_data = (const float *) output_grads->data;
    float *data_grad_data = (float *) data_grad->data;


    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // creating input descriptor
    //hipdnnTensorDescriptor_t input_descriptor;
    //checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    //setTensorDescriptor(input_descriptor, input->ndim, input->shape);

    // creating output_grads descriptor
    hipdnnTensorDescriptor_t output_grads_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_grads_descriptor));
    setTensorDescriptor(output_grads_descriptor, output_grads->ndim, output_grads->shape);

    // create convolution tensor
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
            /*pad_height=*/padding_height,
            /*pad_width=*/padding_width,
            /*vertical_stride=*/stride_height,
            /*horizontal_stride=*/stride_width,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
            /*computeType=*/HIPDNN_DATA_FLOAT));
    // create filter tensors
    hipdnnFilterDescriptor_t filter_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filter_descriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/num_outputs,
            /*in_channels=*/num_filters,
            /*kernel_height=*/filter_height,
            /*kernel_width=*/filter_width));

    // create output descriptor
    //const int output_dim = input_dim;
    //const int output_height = (int)((input->shape[2] - filter_height + 2*padding_height)/stride_height + 1);
    //const int output_width = (int)((input->shape[3] - filter_width + 2*padding_width)/stride_width + 1);
    //const long output_shapes[] = {input->shape[0], num_filters, output_height, output_width};

    // creating output_grads descriptor
    //hipdnnTensorDescriptor_t output_descriptor;
    //checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    //setTensorDescriptor(output_descriptor, output_dim, output_shapes);
    // creating output_grads descriptor
    hipdnnTensorDescriptor_t data_grads_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&data_grads_descriptor));
    setTensorDescriptor(data_grads_descriptor, data_grad->ndim, data_grad->shape);

    hipdnnConvolutionBwdDataAlgo_t backward_data_algo;
    checkCUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnn,
                                                        filter_descriptor,
                                                        output_grads_descriptor,
                                                        convolution_descriptor,
                                                        data_grads_descriptor,
                                                        HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
                                                        0,
                                                        &backward_data_algo));

    size_t workspace_bytes{0};
    checkCUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn,
                                                            filter_descriptor,
                                                            output_grads_descriptor,
                                                            convolution_descriptor,
                                                            data_grads_descriptor,
                                                            backward_data_algo,
                                                            &workspace_bytes));

    //std::cout << "workspace size: " << workspace_bytes << std::endl;

    void *d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    const float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnConvolutionBackwardData(cudnn,
                                            &alpha,
                                            filter_descriptor,
                                            filter_date,
                                            output_grads_descriptor,
                                            output_grads_data,
                                            convolution_descriptor,
                                            backward_data_algo,
                                            d_workspace,
                                            workspace_bytes,
                                            &beta,
                                            data_grads_descriptor,
                                            data_grad_data));

    // Release resources
    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(data_grads_descriptor);
    hipdnnDestroyTensorDescriptor(output_grads_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);

    //std::cout << "leaveing cudnnConv2DBackwardData" << std::endl;
    return 0;

}


int cudnnConv2DBackwardFilter(const DLArrayHandle input,
                              const DLArrayHandle output_grads,
                              const int stride_height,
                              const int stride_width,
                              const int padding_height,
                              const int padding_width,
                              DLArrayHandle filter_grad) {


    const int input_dim = input->ndim;
    const int filter_dim = filter_grad->ndim;
    //const int filter_grad_dim = filter_grad->ndim;
    assert(input_dim == 4);
    assert(filter_dim == 4);
    //assert(filter_grad_dim == filter_dim);

    const int num_filters = filter_grad->shape[0];
    const int num_outputs = filter_grad->shape[1];
    const int filter_height = filter_grad->shape[2];
    const int filter_width = filter_grad->shape[3];

    const float *input_data = (const float *) input->data;
    const float *output_grads_data = (const float *) output_grads->data;
    //const float *filter_date = (const float *) filter->data;
    float *filter_grad_data = (float *) filter_grad->data;

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // creating input descriptor
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    setTensorDescriptor(input_descriptor, input->ndim, input->shape);

    // creating output_grads descriptor
    hipdnnTensorDescriptor_t output_grads_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_grads_descriptor));
    setTensorDescriptor(output_grads_descriptor, output_grads->ndim, output_grads->shape);

    // create convolution tensor
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
            /*pad_height=*/padding_height,
            /*pad_width=*/padding_width,
            /*vertical_stride=*/stride_height,
            /*horizontal_stride=*/stride_width,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/HIPDNN_CROSS_CORRELATION,
            /*computeType=*/HIPDNN_DATA_FLOAT));


    // create filter tensors
    hipdnnFilterDescriptor_t filter_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filter_descriptor,
            /*dataType=*/HIPDNN_DATA_FLOAT,
            /*format=*/HIPDNN_TENSOR_NCHW,
            /*out_channels=*/num_outputs,
            /*in_channels=*/num_filters,
            /*kernel_height=*/filter_height,
            /*kernel_width=*/filter_width));


    hipdnnConvolutionBwdFilterAlgo_t backward_filter_algo;
    checkCUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn,
                                                          input_descriptor,
                                                          output_grads_descriptor,
                                                          convolution_descriptor,
                                                          filter_descriptor,
                                                          HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
                                                          0,
                                                          &backward_filter_algo));

    size_t workspace_bytes{0};
    checkCUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn,
                                                              input_descriptor,
                                                              output_grads_descriptor,
                                                              convolution_descriptor,
                                                              filter_descriptor,
                                                              backward_filter_algo,
                                                              &workspace_bytes));
    void *d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    const float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnConvolutionBackwardFilter(cudnn,
                                              &alpha,
                                              input_descriptor,
                                              input_data,
                                              output_grads_descriptor,
                                              output_grads_data,
                                              convolution_descriptor,
                                              backward_filter_algo,
                                              d_workspace,
                                              workspace_bytes,
                                              &beta,
                                              filter_descriptor,
                                              filter_grad_data));


    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_grads_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);

    return 0;
}


int cudnnPoolForward(const DLArrayHandle input,
                     const int pooling_height,
                     const int pooling_width,
                     const int stride_height,
                     const int stride_width,
                     const char *mode,
                     DLArrayHandle output) {

    //std::cout << mode << std::endl;
    const int input_dim = input->ndim;
    const int output_dim = output->ndim;
    assert(input_dim == 4);
    assert(output_dim == 4);

    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;

    hipdnnPoolingMode_t pooling_mode = HIPDNN_POOLING_MAX;
    std::string str_mode(mode);
    if (str_mode.compare("average") == 0) {
        std::cout << str_mode << std::endl;
        pooling_mode = HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING;
        std::cout << pooling_mode << std::endl;
    }

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // creating input and output tensors
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    setTensorDescriptor(input_descriptor, input->ndim, input->shape);

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    setTensorDescriptor(output_descriptor, output->ndim, output->shape);

    hipdnnPoolingDescriptor_t pooling_descriptor;
    checkCUDNN(hipdnnCreatePoolingDescriptor(&pooling_descriptor));
    checkCUDNN(hipdnnSetPooling2dDescriptor(pooling_descriptor,
                                           pooling_mode,
                                           HIPDNN_PROPAGATE_NAN,
                                           pooling_height,
                                           pooling_width,
                                           0, // TODO: parameterize vertical padding
                                           0, // TODO: parameterize horizontal padding
                                           stride_height,
                                           stride_width));


    const float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnPoolingForward(cudnn,
                                   pooling_descriptor,
                                   &alpha,
                                   input_descriptor,
                                   input_data,
                                   &beta,
                                   output_descriptor,
                                   output_data));

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyPoolingDescriptor(pooling_descriptor);

    hipdnnDestroy(cudnn);
    return 0;
}

int cudnnPoolBackward(const DLArrayHandle input,
                      const DLArrayHandle output_grads,
                      const DLArrayHandle output,
                      const int pooling_height,
                      const int pooling_width,
                      const int stride_height,
                      const int stride_width,
                      DLArrayHandle pool_grad) {

    return 0;
}





