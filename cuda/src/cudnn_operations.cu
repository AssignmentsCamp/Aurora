#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

#define checkCUDNN(expression)                                  \
{                                                               \
    hipdnnStatus_t status = (expression);                        \
    if (status != HIPDNN_STATUS_SUCCESS) {                       \
        std::cerr  << "Error on line " << __LINE__ << ": "      \
                   << hipdnnGetErrorString(status) << std::endl; \
        std::exit(EXIT_FAILURE);                                \
    }                                                           \
}

int setTensorDescriptor(hipdnnTensorDescriptor_t activationDesc,
                           const int numDim,
                           const long shape[]){
    int batchSize = 0;
    int channels = 0;
    switch(numDim){
        case 2:
            batchSize = shape[0];
            channels = shape[1];
            checkCUDNN(hipdnnSetTensor4dDescriptor(activationDesc,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batchSize,
                                          channels, 1, 1));
            break;

        case 4:
            batchSize = shape[0];
            channels = shape[1];
            int height = shape[2];
            int width = shape[3];
            checkCUDNN(hipdnnSetTensor4dDescriptor(activationDesc,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batchSize,
                                          channels,
                                          height,
                                          width));
            break;
        // TODO: handle other cases and errors

    }
    return 0;
}

int cudnnReLUForward(const DLArrayHandle input, DLArrayHandle output) {
	const float *input_data = (const float *) input->data;
	float *output_data = (float *) output->data;

	assert(input->shape[0] == output->shape[0]);
	assert(input->shape[1] == output->shape[1]);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnActivationDescriptor_t activation_descriptor;
    checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
    checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
                                           HIPDNN_ACTIVATION_RELU, // type of activation
                                           HIPDNN_PROPAGATE_NAN, // reluNanOpt
                                           0));  //relu_coef

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    setTensorDescriptor(input_descriptor, input->ndim, input->shape);

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    setTensorDescriptor(output_descriptor, output->ndim, output->shape);

    const float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnActivationForward(cudnn,
                                      activation_descriptor,
                                      &alpha,
                                      input_descriptor,
                                      input_data,
                                      &beta,
                                      output_descriptor,
                                      output_data));

    hipdnnDestroyActivationDescriptor(activation_descriptor);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);

	return 0;
}

int cudnnConv2DForward(const DLArrayHandle input,
                       const DLArrayHandle filter,
                       const DLArrayHandle bias,
                       const int stride_height,
                       const int stride_width,
                       const int padding_height,
                       const int padding_width,
                       DLArrayHandle output){

    const int input_dim = input->ndim;
    const int output_dim = output->ndim;
    assert(input_dim == 4);
    assert(output_dim == 4);

    const int filter_shape = filter->ndim;
    assert(filter_shape == 4);
    const int num_filters = filter->shape[0];
    const int num_outputs = filter->shape[1];
    const int filter_height = filter->shape[2];
    const int filter_width = filter->shape[3];

    const float *input_data = (const float *) input->data;
    const float *filter_date = (const float *) filter->data;
	float *output_data = (float *) output->data;

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);


    // creating input and output tensors
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    setTensorDescriptor(input_descriptor, input->ndim, input->shape);

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    setTensorDescriptor(output_descriptor, output->ndim, output->shape);

    // create filter tensors
    hipdnnFilterDescriptor_t filter_descriptor;
    checkCUDNN(hipdnnCreateFilterDescriptor(&filter_descriptor));
    checkCUDNN(hipdnnSetFilter4dDescriptor(filter_descriptor,
                                        /*dataType=*/HIPDNN_DATA_FLOAT,
                                        /*format=*/HIPDNN_TENSOR_NCHW,
                                        /*out_channels=*/num_outputs,
                                        /*in_channels=*/num_filters,
                                        /*kernel_height=*/filter_height,
                                        /*kernel_width=*/filter_width));
    // create convolution tensor
    hipdnnConvolutionDescriptor_t convolution_descriptor;
    checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
    checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                             /*pad_height=*/padding_height,
                                             /*pad_width=*/padding_width,
                                             /*vertical_stride=*/stride_height,
                                             /*horizontal_stride=*/stride_width,
                                             /*dilation_height=*/1,
                                             /*dilation_width=*/1,
                                             /*mode=*/HIPDNN_CROSS_CORRELATION,
                                             /*computeType=*/HIPDNN_DATA_FLOAT));

    hipdnnConvolutionFwdAlgo_t convolution_algorithm;
    checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn,
                                          input_descriptor,
                                          filter_descriptor,
                                          convolution_descriptor,
                                          output_descriptor,
                                          HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                          /*memoryLimitInBytes=*/0,
                                          &convolution_algorithm));

    size_t workspace_bytes{0};
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn,
                                                     input_descriptor,
                                                     filter_descriptor,
                                                     convolution_descriptor,
                                                     output_descriptor,
                                                     convolution_algorithm,
                                                     &workspace_bytes));
    //std::cerr << "Workspace size: " << (workspace_bytes / 1048576.0) << "MB" << std::endl;
    assert(workspace_bytes > 0);

    void* d_workspace{nullptr};
    hipMalloc(&d_workspace, workspace_bytes);

    const float alpha = 1.0f, beta = 0.0f;

    checkCUDNN(hipdnnConvolutionForward(cudnn,
                                     &alpha,
                                     input_descriptor,
                                     input_data,
                                     filter_descriptor,
                                     filter_date,
                                     convolution_descriptor,
                                     convolution_algorithm,
                                     d_workspace,
                                     workspace_bytes,
                                     &beta,
                                     output_descriptor,
                                     output_data));

    hipFree(d_workspace);

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);

    hipdnnDestroy(cudnn);

    return 0;
}


int cudnnMaxPoolingForward(const DLArrayHandle input,
                        const int pooling_height,
                        const int pooling_width,
                        const int stride_height,
                        const int stride_width,
                        const int mode,
                        DLArrayHandle output){

    const int input_dim = input->ndim;
    const int output_dim = output->ndim;
    assert(input_dim == 4);
    assert(output_dim == 4);

    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // creating input and output tensors
    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    setTensorDescriptor(input_descriptor, input->ndim, input->shape);

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    setTensorDescriptor(output_descriptor, output->ndim, output->shape);

    hipdnnPoolingDescriptor_t pooling_descriptor;
    checkCUDNN(hipdnnCreatePoolingDescriptor(&pooling_descriptor));
    checkCUDNN(hipdnnSetPooling2dDescriptor(pooling_descriptor,
                                HIPDNN_POOLING_MAX,
                                HIPDNN_PROPAGATE_NAN,
                                pooling_height,
                                pooling_width,
                                0, // TODO: parameterize vertical padding
                                0, // TODO: parameterize horizontal padding
                                stride_height,
                                stride_width));


    const float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnPoolingForward(cudnn,
                                   pooling_descriptor,
                                   &alpha,
                                   input_descriptor,
                                   input_data,
                                   &beta,
                                   output_descriptor,
                                   output_data));

    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyPoolingDescriptor(pooling_descriptor);

    hipdnnDestroy(cudnn);
    return 0;
}

