#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hipDNN.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>

#define checkCUDNN(expression)                                  \
{                                                               \
    hipdnnStatus_t status = (expression);                        \
    if (status != HIPDNN_STATUS_SUCCESS) {                       \
        std::cerr  << "Error on line " << __LINE__ << ": "      \
                   << hipdnnGetErrorString(status) << std::endl; \
        std::exit(EXIT_FAILURE);                                \
    }                                                           \
}

int setTensorDescriptor(hipdnnTensorDescriptor_t activationDesc,
                           const int numDim,
                           const long shape[]){
    int batchSize = 0;
    int channels = 0;

    switch(numDim){
        case 2:
            batchSize = shape[0];
            channels = shape[1];
            checkCUDNN(hipdnnSetTensor4dDescriptor(activationDesc,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batchSize,
                                          channels, 1, 1));
            break;

        case 4:
            batchSize = shape[0];
            channels = shape[1];
            int height = shape[2];
            int width = shape[3];
            checkCUDNN(hipdnnCreateTensorDescriptor(&activationDesc));
            checkCUDNN(hipdnnSetTensor4dDescriptor(activationDesc,
                                          HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT,
                                          batchSize,
                                          channels,
                                          height,
                                          width));
            break;
        // TODO: handle other cases and errors

    }
    return 0;
}

int cudnnReLUForward(const DLArrayHandle input, DLArrayHandle output) {
	const float *input_data = (const float *) input->data;
	float *output_data = (float *) output->data;

	assert(input->shape[0] == output->shape[0]);
	assert(input->shape[1] == output->shape[1]);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnActivationDescriptor_t activation_descriptor;
    checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
    checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
                                        /*mode=*/HIPDNN_ACTIVATION_RELU,
                                        /*reluNanOpt=*/HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/0));

    hipdnnTensorDescriptor_t input_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
    setTensorDescriptor(input_descriptor, input->ndim, input->shape);

    hipdnnTensorDescriptor_t output_descriptor;
    checkCUDNN(hipdnnCreateTensorDescriptor(&output_descriptor));
    setTensorDescriptor(output_descriptor, output->ndim, output->shape);

    const float alpha = 1.0f, beta = 0.0f;
    checkCUDNN(hipdnnActivationForward(cudnn,
                                  activation_descriptor,
                                  &alpha,
                                  input_descriptor,
                                  input_data,
                                  &beta,
                                  output_descriptor,
                                  output_data));
    hipdnnDestroyActivationDescriptor(activation_descriptor);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
	return 0;
}

